#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
	 LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
	 http://lammps.sandia.gov, Sandia National Laboratories
	 Steve Plimpton, sjplimp@sandia.gov

	 Copyright (2003) Sandia Corporation.	Under the terms of Contract
	 DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
	 certain rights in this software.	This software is distributed under 
	 the GNU General Public License.

	 See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */
#include "mpi.h"
#include "stdio.h"
#include "string.h"
#include "force.h"
#include "update.h"
#include "respa.h"
#include "error.h"

#include "atom_vec_meso.h"
#include "fix_mui_gpu.h"
#include "engine_meso.h"
#include "atom_meso.h"
#include "comm_meso.h"

#include "atom.h"
#include "comm.h"
#include "input.h"
#include "variable.h"

#include "mui/mui.h"

using namespace LAMMPS_NS;
using namespace FixConst;
using namespace std;

__global__ void gpu_push_gather(
	double4* __restrict push_buffer,
	uint* __restrict push_count,
	r64* __restrict coord_x,
	r64* __restrict coord_y,
	r64* __restrict coord_z,
	r64* __restrict veloc_x,
	r64* __restrict veloc_y,
	r64* __restrict veloc_z,
	int* __restrict mask,
	const r64 send_upper,
	const r64 send_lower,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( ( mask[i] & groupbit ) && coord_z[i] >= send_lower && coord_z[i] <= send_upper ) {
			uint p = atomicInc( push_count, 0xFFFFFFFF );
			double4 info;
			info.x = coord_x[i];
			info.y = coord_y[i];
			info.z = coord_z[i];
			info.w = veloc_x[i];
			push_buffer[p] = info;
		}
	}
}

vector<double4> FixMUIGPU::gpu_push() {
	static int2 grid_cfg;
	static HostScalar<double4> hst_push_buffer(this->lmp,"FixMUI::push_buffer");
	static DeviceScalar<uint>  dev_push_count (this->lmp,"FixMUI::push_count");

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_push_gather, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_push_gather), hipFuncCachePreferL1 );
		dev_push_count.grow(1);
	}
	if ( hst_push_buffer.n_elem() < atom->nlocal ) {
		hst_push_buffer.grow( atom->nlocal );
	}

	dev_push_count.set( 0, meso_device->stream() );
	gpu_push_gather<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_push_buffer,
		dev_push_count,
		meso_atom->dev_coord(0),
		meso_atom->dev_coord(1),
		meso_atom->dev_coord(2),
		meso_atom->dev_veloc(0),
		meso_atom->dev_veloc(1),
		meso_atom->dev_veloc(2),
		meso_atom->dev_mask,
		send_upper,
		send_lower,
		groupbit,
		atom->nlocal );

	uint n;
	dev_push_count.download( &n, 1 );
	meso_device->sync_device();
	vector<double4> result;
	for(int i=0;i<n;i++) result.push_back(hst_push_buffer[i]);
	return result;
}

__global__ void gpu_fetch_pred(
	int* __restrict pred,
	double4* __restrict loc,
	r64* __restrict coord_x,
	r64* __restrict coord_y,
	r64* __restrict coord_z,
	int* __restrict mask,
	const r64 recv_upper,
	const r64 recv_lower,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( ( mask[i] & groupbit ) && coord_z[i] >= recv_lower && coord_z[i] <= recv_upper ) {
			pred[i] = 1;
			loc[i].x = coord_x[i];
			loc[i].y = coord_y[i];
			loc[i].z = coord_z[i];
		}
		else
			pred[i] = 0;
	}
}

pair<vector<int>, vector<double4> > FixMUIGPU::gpu_fetch_predicate() {
	static int2 grid_cfg;
	static HostScalar<int>     hst_fetch_pred(this->lmp,"FixMUI::fetch_pred");
	static HostScalar<double4> hst_fetch_loc(this->lmp,"FixMUI::fetch_coord");
	static vector<int> host_buffer;

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_fetch_pred, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_fetch_pred), hipFuncCachePreferL1 );
	}
	if ( hst_fetch_pred.n_elem() < atom->nlocal ) {
		hst_fetch_pred.grow( atom->nlocal );
		hst_fetch_loc.grow( atom->nlocal );
	}

	gpu_fetch_pred<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_fetch_pred,
		hst_fetch_loc,
		meso_atom->dev_coord(0),
		meso_atom->dev_coord(1),
		meso_atom->dev_coord(2),
		meso_atom->dev_mask,
		recv_upper,
		recv_lower,
		groupbit,
		atom->nlocal );

	meso_device->sync_device();
	vector<int> result_first;
	vector<double4> result_second;
	for(int i=0;i<hst_fetch_pred.n_elem();i++) {
		result_first.push_back( hst_fetch_pred[i] );
		result_second.push_back( hst_fetch_loc[i] );
	}
	return make_pair(result_first,result_second);
}

__global__ void gpu_scatter_fetch(
	int* __restrict pred,
	double* __restrict vres,
	r64* __restrict veloc_x,
	r64* __restrict veloc_y,
	r64* __restrict veloc_z,
	int* __restrict mask,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( pred[i] ) veloc_x[i] += ( vres[i] - veloc_x[i] ) * 1.00;
	}
}

void FixMUIGPU::gpu_fetch( pair<vector<int>, vector<double> > result ) {
	static int2 grid_cfg;
	static HostScalar<int>    hst_pred(this->lmp,"FixMUI::dev_pred");
	static HostScalar<double> hst_vres(this->lmp,"FixMUI::dev_r");

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_scatter_fetch, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_scatter_fetch), hipFuncCachePreferL1 );
	}
	if ( hst_pred.n_elem() < atom->nlocal ) {
		hst_pred.grow( atom->nlocal );
		hst_vres.grow( atom->nlocal );
	}

	for(int i=0;i<result.first.size();i++) {
		hst_pred[i] = result.first[i];
		hst_vres[i] = result.second[i];
	}
	gpu_scatter_fetch<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_pred,
		hst_vres,
		meso_atom->dev_veloc(0),
		meso_atom->dev_veloc(1),
		meso_atom->dev_veloc(2),
		meso_atom->dev_mask,
		groupbit,
		atom->nlocal );
}

mui::point3d point( double4 x ) {
	mui::point3d p;
	p[0] = x.x;
	p[1] = x.y;
	p[2] = x.z;
	return p;
}

FixMUIGPU::FixMUIGPU(LAMMPS *lmp, int narg, char **arg) :
	Fix(lmp, narg, arg),
	MesoPointers(lmp)
{
	// if (narg != 9) error->all(FLERR,"Illegal fix mui arguments");
	interface = new mui::uniface<mui::default_config>( arg[3] );
	send_upper = atof(arg[4]);
	send_lower = atof(arg[5]);
	recv_upper = atof(arg[6]);
	recv_lower = atof(arg[7]);
	sample_rc  = atof(arg[8]);
}

FixMUIGPU::~FixMUIGPU()
{
	if ( interface ) delete interface;
}

int FixMUIGPU::setmask()
{
	int mask = 0;
	mask |= POST_INTEGRATE;
	mask |= END_OF_STEP;
	return mask;
}

void FixMUIGPU::init()
{
}

void FixMUIGPU::post_integrate()
{
	vector<double4> info = gpu_push();

	for (int i = 0; i < info.size(); i++) {
	    // fprintf(screen, "<<<debug gpu push>>> before #%d point loc = %f, %f, %f  velocity = %f\n", info[i].x, info[i].y, info[i].z, info[i].w);
		// fprintf(logfile, "<<<debug gpu push>>> before #%d point loc = %f, %f, %f  velocity = %f\n", info[i].x, info[i].y, info[i].z, info[i].w);
		interface->push( "velocity_x", point(info[i]), info[i].w );
	}

	double time = update->ntimestep * update->dt;
	interface->commit( time );
	interface->barrier( time - 1);
	interface->forget( time - 1 );

}

void FixMUIGPU::end_of_step()
{
	int nlocal = atom->nlocal;

	mui::sampler_shepard_quintic <> quintic(sample_rc);
	mui::temporal_sampler_exact<>       texact(0);

	pair<vector<int>, vector<double4> > pred = gpu_fetch_predicate();
	pair<vector<int>, vector<double> > result;

	double t = update->ntimestep * update->dt;

	// mui::point3d testP;
	// testP[0] = 0.0;
	// testP[1] = 0.0;
	// testP[2] = 0.0;
	
	// double testVal = interface->fetch( "velocity_x", testP, t, quintic, texact );
			
    // fprintf(screen, "<<< GPU debug >>> value is %f at time %f\n", testVal, t);
	// fprintf(logfile, "<<< GPU debug >>> value is %f at time %f\n", testVal, t);

	for (int i = 0; i < nlocal; i++) {
		if ( pred.first[i] ) {
			double res = interface->fetch( "velocity_x", point(pred.second[i]), t, quintic, texact );
			result.second.push_back( res );

			// if (screen)
			// 	fprintf(screen, "<<<debug gpu fetch>>> #%d point loc = %f, %f, %f  velocity = %f\n", pred.second[i].x, pred.second[i].y, pred.second[i].z, res);
			// if (logfile)
			// 	fprintf(logfile, "<<<debug gpu fetch>>> #%d point loc = %f, %f, %f  velocity = %f\n", pred.second[i].x, pred.second[i].y, pred.second[i].z, res);

		} else
			result.second.push_back( 0 );
	}
	result.first = pred.first;

	gpu_fetch( result );
}

